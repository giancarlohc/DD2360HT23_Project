#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <iostream>
#include <cmath>
#include <sys/time.h>


#define CheckCudaError(stmt)                                               \
  do {                                                               \
      hipError_t err = stmt;                                        \
      if (err != hipSuccess) {                                      \
          printf("ERROR. Failed to run stmt %s\n", #stmt);           \
          break;                                                     \
      }                                                              \
  } while (0)

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                                        \
  do {                                                                                           \
      hipsolverStatus_t err_ = (err);                                                             \
      if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                                     \
          printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__);                      \
          throw std::runtime_error("cusolver error");                                            \
      }                                                                                          \
  } while (0)


// Timer setup
#include <sys/time.h>
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// Initialize matrix from file input
void InitMat(FILE* fp, double* ary, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (fscanf(fp, "%lf", &ary[size * j + i]) != 1) {
                perror("Failed to read the matrix");
                exit(1);
            }
        }
    }
}

// Initialize array from file input
void InitAry(FILE* fp, double* ary, int size) {
    for (int i = 0; i < size; i++) {
        if (fscanf(fp, "%lf", &ary[i]) != 1) {
            perror("Failed to read the array");
            exit(1);
        }
    }
}

void PrintMat(double* ary, int size)
{
    int i, j;

    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            printf("%8.2f ", *(ary + size * i + j));
        }
        printf("\n");
    }
    printf("\n");
}

void PrintAry(double* ary, int size)
{
    for (int i = 0; i < size; i++) {
        printf("%8.2f ", *(ary + i));
    }
}

void InitProblemOnce(char* filename, int* size, double** a, double** b, double** slnVec, double** X) {
    FILE* fp = fopen(filename, "r");
    if (!fp) {
        perror("Unable to open the file");
        exit(1);
    }

    printf("Read input from is: %s\n", filename);

    if (fscanf(fp, "%d", size) != 1) {
        perror("Failed to read the size");
        exit(1);
    }
    printf("The input matrix A's size is: %d\n", *size);

    // Allocate memory for a and b
    hipMallocManaged(a, *size * *size * sizeof(double));
    hipMallocManaged(b, *size * sizeof(double));

    // Advise the CUDA memory manager to set the preferred location for a and b
    hipMemAdvise(a, *size * *size * sizeof(double), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, *size * sizeof(double), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);

    InitMat(fp, *a, *size);
    InitAry(fp, *b, *size);
    *slnVec = (double*)malloc((*size) * sizeof(double));
    InitAry(fp, *slnVec, *size);
}

int main(int argc, char* argv[]) {
    //begin timing
    struct timeval time_start;
    gettimeofday(&time_start, NULL);

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    int size;
    double* A, * B, * slnVec, * X;
    int id = hipGetDevice(&id);

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <filename>\n";
        exit(1);
    }

    char* filename = argv[1];

    InitProblemOnce(filename, &size, &A, &B, &slnVec, &X);

    // PrintMat(A, size);
    // PrintAry(B, size);
    // printf("\n");

    // printf("Solution: ");
    // PrintAry(slnVec, size);
    // printf("\n");

    // Initialize cuSolver
    hipsolverDnCreate(&cusolverH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);

    // Allocate memory for devIpiv and devInfo
    int* devIpiv, * devInfo;
    hipMallocManaged(&devIpiv, size * sizeof(int));
    hipMallocManaged(&devInfo, sizeof(int));

    // Advise the CUDA memory manager to set the preferred location for devIpiv, devInfo, A and B
    hipMemAdvise(devIpiv, size * sizeof(int), hipMemAdviseSetPreferredLocation, id);
    hipMemAdvise(devInfo, sizeof(int), hipMemAdviseSetPreferredLocation, id);
    hipMemAdvise(A, size * size * sizeof(double), hipMemAdviseSetPreferredLocation, id);
    hipMemAdvise(B, size * sizeof(double), hipMemAdviseSetPreferredLocation, id);

    // Asynchronously prefetch A and B to GPU
    hipMemPrefetchAsync(A, size * size * sizeof(double), id);
    hipMemPrefetchAsync(B, size * sizeof(double), id);

    // Allocate memory for workspace
    double* workspace;
    int workspace_size;
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, size, size, A, size, &workspace_size));
    hipDeviceSynchronize();
    hipMallocManaged(&workspace, workspace_size * sizeof(double));
    hipMemAdvise(workspace, workspace_size * sizeof(double), hipMemAdviseSetPreferredLocation, id);

    // LU factorization
    double start = cpuSecond();
    CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, size, size, A, size, workspace, devIpiv, devInfo));
    hipDeviceSynchronize();
    double stop = cpuSecond();
    printf("Time for LU factorization: %f sec\n", stop - start);

    // Solve Ax = B
    start = cpuSecond();
    CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, size, 1, A, size, devIpiv, B, size, devInfo));
    hipDeviceSynchronize();
    stop = cpuSecond();
    printf("Time for solving Ax = B: %f sec\n", stop - start);

    // Asynchronously prefetch B to CPU
    hipMemPrefetchAsync(B, size * sizeof(double), hipCpuDeviceId);

    // Print solution
    // printf("Solution:\n");
    bool isCorrect = true;
    for (int i = 0; i < size; i++) {
        // printf("%f ", B[i]);
        if (fabs(B[i] - slnVec[i]) > 1e-5) {
            isCorrect = false;
            // break;
        }
    }

    printf("\n");
    std::cout << "Results are " << (isCorrect ? "correct" : "incorrect") << std::endl;

    // Cleanup
    hipFree(A);
    hipFree(B);
    free(slnVec);
    hipFree(workspace);
    hipFree(devIpiv);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);

    //end timing
    struct timeval time_end;
    gettimeofday(&time_end, NULL);
    unsigned int time_total = (time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec);
    printf("\nTime total (including memory transfers)\t%f sec\n", time_total * 1e-6);

    return 0;
}

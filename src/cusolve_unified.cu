#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <iostream>
#include <cmath>
#include <sys/time.h>


#define CheckCudaError(stmt)                                               \
  do {                                                               \
      hipError_t err = stmt;                                        \
      if (err != hipSuccess) {                                      \
          printf("ERROR. Failed to run stmt %s\n", #stmt);           \
          break;                                                     \
      }                                                              \
  } while (0)

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                                        \
  do {                                                                                           \
      hipsolverStatus_t err_ = (err);                                                             \
      if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                                     \
          printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__);                      \
          throw std::runtime_error("cusolver error");                                            \
      }                                                                                          \
  } while (0)


// Timer setup
#include <sys/time.h>
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// Initialize matrix from file input
void InitMat(FILE* fp, double* ary, int size) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (fscanf(fp, "%lf", &ary[size * j + i]) != 1) {
                perror("Failed to read the matrix");
                exit(1);
            }
        }
    }
}

// Initialize array from file input
void InitAry(FILE* fp, double* ary, int size) {
    for (int i = 0; i < size; i++) {
        if (fscanf(fp, "%lf", &ary[i]) != 1) {
            perror("Failed to read the array");
            exit(1);
        }
    }
}

void PrintMat(double* ary, int size)
{
    int i, j;

    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            printf("%8.2f ", *(ary + size * i + j));
        }
        printf("\n");
    }
    printf("\n");
}

void PrintAry(double* ary, int size)
{
    for (int i = 0; i < size; i++) {
        printf("%8.2f ", *(ary + i));
    }
}

void InitProblemOnce(char* filename, int* size, double** a, double** b, double** slnVec, double** X) {
    FILE* fp = fopen(filename, "r");
    if (!fp) {
        perror("Unable to open the file");
        exit(1);
    }

    printf("Read input from is: %s\n", filename);

    if (fscanf(fp, "%d", size) != 1) {
        perror("Failed to read the size");
        exit(1);
    }
    printf("The input matrix A's size is: %d\n", *size);

    hipMallocManaged(a, *size * *size * sizeof(double));
    hipMallocManaged(b, *size * sizeof(double));

    hipMemAdvise(a, *size * *size * sizeof(double), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, *size * sizeof(double), hipMemAdviseSetPreferredLocation, hipCpuDeviceId);

    // *a = (double*)malloc((*size) * (*size) * sizeof(double));
    InitMat(fp, *a, *size);

    // *b = (double*)malloc((*size) * sizeof(double));
    InitAry(fp, *b, *size);

    *slnVec = (double*)malloc((*size) * sizeof(double));
    InitAry(fp, *slnVec, *size);

    // *X = (double*)malloc((*size) * sizeof(double));
}

int main(int argc, char* argv[]) {
    //begin timing
    struct timeval time_start;
    gettimeofday(&time_start, NULL);	

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    int size;
    double* A, * B, * slnVec, * X;
    int id = hipGetDevice(&id);

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <filename>\n";
        exit(1);
    }

    char* filename = argv[1];

    InitProblemOnce(filename, &size, &A, &B, &slnVec, &X);

    // PrintMat(A, size);
    // PrintAry(B, size);
    // printf("\n");

    // printf("Solution: ");
    // PrintAry(slnVec, size);
    // printf("\n");

    // Initialize cuSolver
    hipsolverDnCreate(&cusolverH);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsolverSetStream(cusolverH, stream);

    // Device memory management
    // double* d_A = NULL, * d_B = NULL;
    // CheckCudaError(hipMalloc((void**)&d_A, sizeof(double) * size * size));
    // CheckCudaError(hipMalloc((void**)&d_B, sizeof(double) * size));


    int* devIpiv, * devInfo;
    hipMallocManaged(&devIpiv, size * sizeof(int));
    hipMallocManaged(&devInfo, sizeof(int));
    hipMemAdvise(devIpiv, size * sizeof(int), hipMemAdviseSetPreferredLocation, id);
    hipMemAdvise(devInfo, sizeof(int), hipMemAdviseSetPreferredLocation, id);

    // CheckCudaError(hipMalloc((void**)&devIpiv, size * sizeof(int)));
    // CheckCudaError(hipMalloc((void**)&devInfo, sizeof(int)));

    // Copy matrices from host to device
    // CheckCudaError(hipMemcpy(d_A, A, sizeof(double) * size * size, hipMemcpyHostToDevice));
    // CheckCudaError(hipMemcpy(d_B, B, sizeof(double) * size, hipMemcpyHostToDevice));
     
    hipMemAdvise(A, size * size * sizeof(double), hipMemAdviseSetPreferredLocation, id);
    hipMemAdvise(B, size * sizeof(double), hipMemAdviseSetPreferredLocation, id);

    hipMemPrefetchAsync(A, size * size * sizeof(double), id);
    hipMemPrefetchAsync(B, size * sizeof(double), id);  




    // Allocate workspace
    double* workspace; 
    int workspace_size;
    CUSOLVER_CHECK(hipsolverDnDgetrf_bufferSize(cusolverH, size, size, A, size, &workspace_size));
    hipDeviceSynchronize();
    hipMallocManaged(&workspace, workspace_size * sizeof(double));
    hipMemAdvise(workspace, workspace_size * sizeof(double), hipMemAdviseSetPreferredLocation, id);

    // CheckCudaError(hipMalloc(&workspace, workspace_size * sizeof(double)));
    // printf("%d", workspace_size);

    // LU factorization
    double start = cpuSecond();
    CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, size, size, A, size, workspace, devIpiv, devInfo));
    hipDeviceSynchronize();
    double stop = cpuSecond();
    printf("Time for LU factorization: %f sec\n", stop - start);
    
    // int* hIpiv;
    // hIpiv = (int*)malloc(size * sizeof(int));
    // hipMemcpy(hIpiv, devIpiv, size * sizeof(int), hipMemcpyDeviceToHost);
    // CheckCudaError(hipMemcpy(A, d_A, sizeof(double) * size * size, hipMemcpyDeviceToHost));
    
    // PrintMat(A, size); 
    // for(int i = 0; i < size; i++)
    // {
    //   printf("%d ", *(hIpiv + i));
    // }
    printf("\n");
    
    // Solve Ax = B
    start = cpuSecond();
    CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, size, 1, A, size, devIpiv, B, size, devInfo));
    hipDeviceSynchronize();
    stop = cpuSecond();
    printf("Time for solving Ax = B: %f sec\n", stop - start);

    // Copy result back to host
    // CheckCudaError(hipMemcpy(X, d_B, sizeof(double) * size, hipMemcpyDeviceToHost));
    hipMemPrefetchAsync(B, size * sizeof(double), hipCpuDeviceId);  

    // Print solution
    // printf("Solution:\n");
    bool isCorrect = true;
    for (int i = 0; i < size; i++) {
        // printf("%f ", B[i]);
        if (fabs(B[i] - slnVec[i]) > 1e-5) {
            isCorrect = false;
            // break;
        }
    }

    printf("\n");
    std::cout << "Results are " << (isCorrect ? "correct" : "incorrect") << std::endl;

    // Cleanup
    hipFree(A);
    hipFree(B);
    free(slnVec);
    // hipFree(X);
    // hipFree(d_A);
    // hipFree(d_B);
    hipFree(workspace);
    hipFree(devIpiv);
    hipFree(devInfo);
    hipsolverDnDestroy(cusolverH);

    //end timing
    struct timeval time_end;
    gettimeofday(&time_end, NULL);
    unsigned int time_total = (time_end.tv_sec * 1000000 + time_end.tv_usec) - (time_start.tv_sec * 1000000 + time_start.tv_usec);
    printf("\nTime total (including memory transfers)\t%f sec\n", time_total * 1e-6);

    return 0;
}